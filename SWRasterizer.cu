#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>

#include <string>
#include <iostream>
#include <fstream>
#include <vector>

#include "BasicModel.h"
#include "Model.h"
#include "Triangle.h"

// Window (screen) dimensions
#define WindowWidth 2000
#define WindowHeight 2000

// World coordinates bounding box
#define XMinWorld -1
#define XMaxWorld 1
#define YMinWorld -1
#define YMaxWorld 1

// Camera is at origin looking down negative Z, so further away = smaller Z
#define MinZ -10000

#define BLOCK_WIDTH 32

using namespace std;

void init();
void test();
Triangle convertTriTo2D(Triangle);
Vector3 convertVertexTo2D(Vector3);
__device__ __host__ void rasterizeTriangle(Triangle t, float *r, float *g, float *b, float *z);
VectorThree barycentricCoords(Vector3, Vector3, Vector3, VectorThree, float);
void WriteTga(char* outfile);
Vector3 diffuseShadeVertex(Vector3, Vector3);
__global__ void Rasterize(Triangle *d_tris, float *d_zbuf, float *d_red, float *d_green, float *d_blue);
void processTriangles(BasicModel*, float*, float*, float*, float*, bool);

float zbuffer[WindowWidth][WindowHeight];
float red[WindowWidth][WindowHeight];
float green[WindowWidth][WindowHeight];
float blue[WindowWidth][WindowHeight];
Vector3 directionToLight;
Vector3 lightColor;

int main(int argc, char** argv)
{
	bool tileBunnies = false;
	bool useCUDA = false;

	// -t --> make an image with 25 tiled bunnies. else draw just one bunny.
	// -c --> run with CUDA. else run on CPU.
	for (int i = 0; i < argc; ++i)
	{
		if (strcmp("-t", argv[i]) == 0) tileBunnies = true;
		else if (strcmp("-c", argv[i]) == 0) useCUDA = true;
	}

	init();
	
	float xOffsets[5] = {-0.66, -0.33, 0, 0.33, 0.66};
	float yOffsets[5] = {-0.66, -0.33, 0, 0.33, 0.66};
	int scaleFactor;
	
	//Future pointer to device memory for triangle array
   //Triangle *d_tris;
   //Pointers to device memory for rgb and zbuffer arrays
   float *d_zbuf, *d_red, *d_green, *d_blue;

	// Parse the model file
	string filename = argv[1];
	BasicModel* model = new BasicModel(filename);
	
	int arrSize = model->TriangleStructs.size();
	int a2 = WindowWidth*WindowHeight*sizeof(float);

	if (useCUDA)
	{
		//Allocate memory on device for zbuffer and RGB
		hipMalloc((void **)&d_zbuf, a2);
		hipMemset(d_zbuf, MinZ, a2);
		hipMalloc((void **)&d_red, a2);
		hipMemset(d_red, 0, a2);
		hipMalloc((void **)&d_green, a2);
		hipMemset(d_green, 0, a2);
		hipMalloc((void **)&d_blue, a2);
		hipMemset(d_blue, 1, a2);
	}
	
	if (tileBunnies)
	{
		scaleFactor = 3;
		for (int yIndex = 0; yIndex < 5; ++yIndex)
		{
			for (int xIndex = 0; xIndex < 5; ++xIndex)
			{
				model->createTriangleStructs(xOffsets[xIndex], yOffsets[yIndex], scaleFactor);

				processTriangles(model, d_zbuf, d_red, d_green, d_blue, useCUDA);
			}
		}
	}
	else
	{
		scaleFactor = 10;
		model->createTriangleStructs(0, 0, scaleFactor);

		processTriangles(model, d_zbuf, d_red, d_green, d_blue, useCUDA);
	}
	
	// Copy color buffers back to host memory
	for(int i = 0; i < WindowHeight; i++)
	{
		hipMemcpy(red[i], d_red+(i*WindowWidth), WindowWidth*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(green[i], d_green+(i*WindowWidth), WindowWidth*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(blue[i], d_blue+(i*WindowWidth), WindowWidth*sizeof(float), hipMemcpyDeviceToHost);
	}
	
	/*
	// Make an array of our Triangle structs
	Triangle* tris = new Triangle[model.TriangleStructs.size()];
    
	for (int i = 0; i < model.TriangleStructs.size(); ++i)
	{
		tris[i] = model.TriangleStructs[i];

		// do diffuse shading on the vertices. These calculated colors will be
		// linearly interpolated during rasterization.
		tris[i].v1.rgb = diffuseShadeVertex(tris[i].normal, tris[i].v1.rgb);
		tris[i].v2.rgb = diffuseShadeVertex(tris[i].normal, tris[i].v2.rgb);
		tris[i].v3.rgb = diffuseShadeVertex(tris[i].normal, tris[i].v3.rgb);
	}
	
	printf("tris[45]: %lf, %lf\n", tris[14].v1.position.x, tris[12].v1.position.x);

	
	//Allocate memory on device for triangles and copy array over
	hipMalloc((void **)&d_tris, arrSize*sizeof(Triangle));
   hipMemcpy(d_tris, tris, arrSize*sizeof(Triangle), hipMemcpyHostToDevice);
   
   printf("tris[45]: %lf, %lf\n", tris[14].v1.position.x, tris[12].v1.position.x);
   //hipMemcpy(tris, d_tris, arrSize*sizeof(Triangle), hipMemcpyDeviceToHost);
   printf("tris[45]: %lf, %lf\n", tris[45].v1.position.x, tris[35].v1.position.x);

   // rasterize on GPU  (arrSize%BLOCK_WIDTH ? 0 : 1)
   printf("Not Rasterized\n");
   Rasterize<<< arrSize/BLOCK_WIDTH+1, BLOCK_WIDTH >>>(d_tris, d_zbuf, d_red, d_green, d_blue);
   
   printf("Rasterized\n");
   for(int i = 0; i < WindowHeight; i++)
   {
      hipMemcpy(red[i], d_red+(i*WindowWidth), WindowWidth*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(green[i], d_green+(i*WindowWidth), WindowWidth*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(blue[i], d_blue+(i*WindowWidth), WindowWidth*sizeof(float), hipMemcpyDeviceToHost);
   }
   
   //hipMemcpy(tris, d_tris, arrSize*sizeof(Triangle), hipMemcpyDeviceToHost);

   //#ifdef WRITE
   printf("r: %f, g: %f, b: %f\n", red[346][234], green[234][12], blue[292][392]);
   //#endif
   */
	// rasterize each triangle
	/*for (int i = 0; i < arrSize; ++i)
	{
		rasterizeTriangle(convertTriTo2D(tris[i]), *red, *green, *blue, *zbuffer);
	}*/
	

	// Output the image
	printf("write\n");
	WriteTga("image.tga");

	return 0;
}

void processTriangles(BasicModel* model, float* d_zbuf, float* d_red, float* d_green, float* d_blue, bool useCUDA)
{
	//Pointer to device memory for triangle array
	Triangle *d_tris;

	int arrSize = model->TriangleStructs.size();
	
	// Make an array of our Triangle structs
	Triangle* tris = new Triangle[model->TriangleStructs.size()];
	for (int i = 0; i < model->TriangleStructs.size(); ++i)
	{
		tris[i] = model->TriangleStructs[i];

		// do diffuse shading on the vertices. These calculated colors will be
		// linearly interpolated during rasterization.
		tris[i].v1.rgb = diffuseShadeVertex(tris[i].normal, tris[i].v1.rgb);
		tris[i].v2.rgb = diffuseShadeVertex(tris[i].normal, tris[i].v2.rgb);
		tris[i].v3.rgb = diffuseShadeVertex(tris[i].normal, tris[i].v3.rgb);
		
		if (!useCUDA)
		{
			rasterizeTriangle(convertTriTo2D(tris[i]), *red, *green, *blue, *zbuffer);
		}
	}
	
	if (useCUDA)
	{
		//Allocate memory on device for triangles and copy array over
		hipMalloc((void **)&d_tris, arrSize*sizeof(Triangle));
		hipMemcpy(d_tris, tris, arrSize*sizeof(Triangle), hipMemcpyHostToDevice);

		// rasterize on GPU  (arrSize%BLOCK_WIDTH ? 0 : 1)
		printf("Not Rasterized\n");
		Rasterize<<< arrSize/BLOCK_WIDTH+1, BLOCK_WIDTH >>>(d_tris, d_zbuf, d_red, d_green, d_blue);
		printf("Rasterized\n");

		hipFree(d_tris);
	}
	delete tris;
}

void init()
{
	for (int i = 0; i < WindowWidth; ++i)
	{
		for (int j = 0; j < WindowHeight; ++j)
		{
			zbuffer[i][j] = MinZ;
			red[i][j] = 0;
			green[i][j] = 0;
			blue[i][j] = 1;
		}
	}

	// set light to always come from positive Z
	directionToLight.x = 0;
	directionToLight.y = 0;
	directionToLight.z = 1;

	// white light
	lightColor.x = 1;
	lightColor.y = 1;
	lightColor.z = 1;
}

/*
* Calculates colors (RGB) for a vertex using diffuse reflectance.
*
* normal: The vertex's normal vector
* diffuseReflectance: how much diffuse light the vertex reflects (red, green, and blue components)
*
* returns: The vertex's diffuse color represented as an RGB triplet.
*/
Vector3 diffuseShadeVertex(Vector3 normal, Vector3 diffuseReflectance)
{
	Vector3 diffuseColor;
	float nDotL = normal.dotP(directionToLight);

	// red
	diffuseColor.x = diffuseReflectance.x * nDotL * lightColor.x;

	// green
	diffuseColor.y = diffuseReflectance.y * nDotL * lightColor.y;

	// blue
	diffuseColor.z = diffuseReflectance.z * nDotL * lightColor.z;

	return diffuseColor;
}

/*
* Given a Triangle with vertices specified in world coordinates,
* convert the triangle to 2D (screen) coordinates.
*
* t: The triangle in world coordinates
* 
* returns: The triangle converted to screen coordinates
*/
__device__ __host__ Triangle convertTriTo2D(Triangle t)
{
	Triangle converted = t;

	// convert the vertices to screen space
	Vector3 convertedV1 = convertVertexTo2D(t.v1.position);
	Vector3 convertedV2 = convertVertexTo2D(t.v2.position);
	Vector3 convertedV3 = convertVertexTo2D(t.v3.position);
	converted.v1.position = convertedV1;
	converted.v2.position = convertedV2;
	converted.v3.position = convertedV3;

	// calculate bounding box for the triangle (in screen coordinates)
	converted.minX = min(convertedV1.x, min(convertedV2.x, convertedV3.x));
	converted.maxX = max(convertedV1.x, max(convertedV2.x, convertedV3.x));
	converted.minY = min(convertedV1.y, min(convertedV2.y, convertedV3.y));
	converted.maxY = max(convertedV1.y, max(convertedV2.y, convertedV3.y));

	return converted;
}

/*
* Convert the provided point from world coordinates to screen coordinates.
*
* coords: The vertex coordinates we want to convert
* 
* returns: The vertex converted to screen coordinates
*/
__device__ __host__ Vector3 convertVertexTo2D(Vector3 coords)
{
	coords.x = ((coords.x - XMinWorld) * WindowWidth) / (XMaxWorld - XMinWorld);
	coords.y = ((coords.y - YMinWorld) * WindowHeight) / (YMaxWorld - YMinWorld);
	
	return coords;
}

/*
* Rasterize a triangle.
*
* t: The triangle to rasterize (should already be converted to screen coordinates)
*/
__device__ __host__ void rasterizeTriangle(Triangle t, float *r, float *g, float *b, float *z)
{
	Vector3 v1Color = t.v1.rgb;
	Vector3 v2Color = t.v2.rgb;
	Vector3 v3Color = t.v3.rgb;
	float v1Z = t.v1.position.z;
	float v2Z = t.v2.position.z;
	float v3Z = t.v3.position.z;

	// denominator for barycentric coords calculation = (v1.x*v2.y) - (v1.x*v3.y) - (v2.x*v1.y) + (v2.x*v3.y) + (v3.x*v1.y) - (v3.x*v2.y)
	// calculate this once for the triangle
	float denom = (t.v1.position.x * t.v2.position.y) -
		(t.v1.position.x * t.v3.position.y) -
		(t.v2.position.x * t.v1.position.y) +
		(t.v2.position.x * t.v3.position.y) +
		(t.v3.position.x * t.v1.position.y) -
		(t.v3.position.x * t.v2.position.y);
	
	// iterate over each point (pixel) in the triangle's bounding box
	for (int x = t.minX; x < t.maxX; ++x)
	{
		for (int y = t.minY; y < t.maxY; ++y)
		{
			if (x < 0 || x >= WindowWidth || y < 0 || y >= WindowHeight)
				continue;
		
			Vertex2 p;
			p.position.x = x;
			p.position.y = y;

			// get barycentric coordinates for p (the current X/Y position)
			// See utils.h for VectorThree
			VectorThree baryCoords = barycentricCoords(t.v1.position, t.v2.position, t.v3.position, p.position, denom);

			// Test the pixel to see if it's inside the triangle. All three
			// barycentric coordinates must be between 0 and 1 for the pixel
			// to be inside the triangle.
			if (baryCoords.x > 0 && baryCoords.x < 1		// check alpha
				&& baryCoords.y > 0 && baryCoords.y < 1		// check beta
				&& baryCoords.z > 0 && baryCoords.z < 1)	// check gamma
			{
				// linearly interpolate the point's color
				p.rgb.x = baryCoords.x*v1Color.x + baryCoords.y*v2Color.x + baryCoords.z*v3Color.x;	// red
				p.rgb.y = baryCoords.x*v1Color.y + baryCoords.y*v2Color.y + baryCoords.z*v3Color.y;	// green
				p.rgb.z = baryCoords.x*v1Color.z + baryCoords.y*v3Color.z + baryCoords.z*v3Color.z;	// blue

				// linearly interpolate the point's depth
				p.position.z = baryCoords.x*v1Z + baryCoords.y*v2Z + baryCoords.z*v3Z;

				// Z buffer test.
				// The camera is at the origin (0, 0, 0) looking down the negative Z axis.
				// This means closer to the camera = greater Z value.
				if (p.position.z > z[x*WindowWidth+y])
				{
					// write the pixel's color components to our color arrays
					r[x*WindowWidth+y] = p.rgb.x;
					g[x*WindowWidth+y] = p.rgb.y;
					b[x*WindowWidth+y] = p.rgb.z;

					// update the Z buffer
					z[x*WindowWidth+y] = p.position.z;
				}
			}
		}
	}
}

/*
* Calculate the barycentric coordinates for a point with respect to the provided
* vertex positions.
*
* v1, v2, v3: The triangle's vertices
* p: The point to get the barycentric coordinates for (may be inside or outside the triangle)
*
* returns: p's corresponding barycentric coordinates as a Vector3 struct. (x = alpha, y = beta, z = gamma)
*/
__device__ __host__ VectorThree barycentricCoords(Vector3 v1, Vector3 v2, Vector3 v3, VectorThree p, float denom)
{
	// NOTE: these formulas found at http://crackthecode.us/barycentric/barycentric_coordinates.html
	//float denom = (v1.x*v2.y) - (v1.x*v3.y) - (v2.x*v1.y) + (v2.x*v3.y) + (v3.x*v1.y) - (v3.x*v2.y);

	// ((X4 * Y2) - (X4 * Y3) - (X2 * Y4) + (X2 * Y3) + (X3 * Y4) - (X3 * Y2)) 
	float alpha = ((p.x*v2.y) - (p.x*v3.y) - (v2.x*p.y) + (v2.x*v3.y) + (v3.x*p.y) - (v3.x*v2.y)) / denom;

	// ((X1 * Y4) - (X1 * Y3) - (X4 * Y1) + (X4 * Y3) + (X3 * Y1) - (X3 * Y4)) 
	float beta = ((v1.x*p.y) - (v1.x*v3.y) - (p.x*v1.y) + (p.x*v3.y) + (v3.x*v1.y) - (v3.x*p.y)) / denom;

	// ((X1 * Y2) - (X1 * Y4) - (X2 * Y1) + (X2 * Y4) + (X4 * Y1) - (X4 * Y2))
	float gamma = ((v1.x*v2.y) - (v1.x*p.y) - (v2.x*v1.y) + (v2.x*p.y) + (p.x*v1.y) - (p.x*v2.y)) / denom;

   // See utils.h for VectorThree
	VectorThree baryCoords;
	baryCoords.x = alpha;
	baryCoords.y = beta;
	baryCoords.z = gamma;

	return baryCoords;
}

void WriteTga(char *outfile)
{
    FILE *fp = fopen(outfile, "wb"); // originally was just "w"
    if (fp == NULL)
    {
        perror("ERROR: Image::WriteTga() failed to open file for writing!\n");
        exit(EXIT_FAILURE);
    }
    
    // write 24-bit uncompressed targa header
    // thanks to Paul Bourke (http://local.wasp.uwa.edu.au/~pbourke/dataformats/tga/)
    putc(0, fp);
    putc(0, fp);
    
    putc(2, fp); // type is uncompressed RGB
    
    putc(0, fp);
    putc(0, fp);
    putc(0, fp);
    putc(0, fp);
    putc(0, fp);
    
    putc(0, fp); // x origin, low byte
    putc(0, fp); // x origin, high byte
    
    putc(0, fp); // y origin, low byte
    putc(0, fp); // y origin, high byte

    putc(WindowWidth & 0xff, fp); // width, low byte
    putc((WindowWidth & 0xff00) >> 8, fp); // width, high byte

    putc(WindowHeight & 0xff, fp); // height, low byte
    putc((WindowHeight & 0xff00) >> 8, fp); // height, high byte

    putc(24, fp); // 24-bit color depth

    putc(0, fp);

    // write the raw pixel data in groups of 3 bytes (BGR order)
    for (int y = 0; y < WindowHeight; y++)
    {
        for (int x = 0; x < WindowWidth; x++)
        {
            // if color scaling is on, scale 0.0 -> _max as a 0 -> 255 unsigned byte
            unsigned char rbyte, gbyte, bbyte;
            double r = (red[x][y] > 1.0) ? 1.0 : red[x][y];
            double g = (green[x][y] > 1.0) ? 1.0 : green[x][y];
            double b = (blue[x][y] > 1.0) ? 1.0 : blue[x][y];
            rbyte = (unsigned char)(r * 255);
            gbyte = (unsigned char)(g * 255);
            bbyte = (unsigned char)(b * 255);

            putc(bbyte, fp);
            putc(gbyte, fp);
            putc(rbyte, fp);
        }
    }

    fclose(fp);
}

__global__ void Rasterize(Triangle *d_tris, float *d_zbuf, float *d_red, float *d_green, float *d_blue)
{
   int idx = blockIdx.x*BLOCK_WIDTH+threadIdx.x;
   
   rasterizeTriangle(convertTriTo2D(d_tris[idx]), d_red, d_green, d_blue, d_zbuf);
}
